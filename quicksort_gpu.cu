#include "hip/hip_runtime.h"
#include "quicksort_gpu.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <chrono>
#include <algorithm>


template <typename T>
__device__
void swap_d(T& lhs, T& rhs){
    T tmp = lhs;
    lhs = rhs;
    rhs = tmp;
}

template <typename T>
__device__
void selection_sort_gpu(T* d, int low, int high){
    for( auto i = low; i <= high; ++i){
        auto min_val = d[i];
        auto min_idx = i;

        for( int j = i + 1; j <= high; ++j ){
            auto val_j = d[j];

            if( val_j < min_val ){
                min_idx = j;
                min_val = val_j;
            }
        }
        swap_d(d[i], d[min_idx]);
    }
}

template <typename T>
__device__
std::pair<int,int> partition_gpu( T* d, int low, int high){
    // Take pivot in center
    auto pivot = d[(high + low) >> 1];

    while( low < high ){
        // Move low index up while low value is less than pivot
        while( d[low] < pivot ){
            ++low;
        }
        // Move high index down while high value is greater than pivot
        while( d[high] > pivot ){
            --high;
        }

        // Swap points are valid, do the swap!???
        if( low <= high ){
            swap_d(d[low++],d[high--]);
        }

    }

    return std::make_pair(low, high);
}

constexpr int MAX_DEPTH = 24;
constexpr int MIN_SIZE = 1024; // 32 gets us about 10xcpu, 1024 gets about 3xcpu

template <typename T>
__global__
void quicksort_gpu_dyn_worker( T* d, int low, int high, int level ){
    if( high <= low ) return;
    if( level >= MAX_DEPTH || (high - low) < MIN_SIZE ){
        selection_sort_gpu(d, low, high);
        return;
    }

    auto lh = partition_gpu( d, low, high );

    hipStream_t s_l, s_h;
    hipStreamCreateWithFlags(&s_l, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&s_h, hipStreamNonBlocking);
    quicksort_gpu_dyn_worker<<<1, 1, 0, s_l>>>(d, low, lh.second, level + 1);
    quicksort_gpu_dyn_worker<<<1, 1, 0, s_h>>>(d, lh.first, high, level + 1 );

    hipStreamDestroy(s_l);
    hipStreamDestroy(s_h);
}

template <typename T>
__host__
void quicksort_gpu_dyn(std::vector<T> &list){
    // Detect settings
    // Copy to device
    T *d{};
    auto err = hipMalloc((void**)&d, sizeof(T) * list.size() );
    if( err != hipSuccess ) {
        std::cout << "CUDA ERROR Malloc" << err << std::endl;
        return;
    }
    err = hipMemcpy((void**)d, list.data(), sizeof(T) * list.size(), hipMemcpyHostToDevice );
    if( err != hipSuccess ) {
        std::cout << "CUDA ERROR Memcpy->Device" << err << std::endl;
        hipFree(d);
        return;
    }

    err = hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH);
    if( err != hipSuccess ){
        std::cout << "CUDA ERROR set cudaLimitDevRuntimeSyncDepth" << std::endl;
        return;
    }

    quicksort_gpu_dyn_worker<<<1,1>>>(d, 0, list.size() - 1, 0);
    hipDeviceSynchronize();

    err = hipMemcpy(list.data(), (void**)d, sizeof(T) * list.size(), hipMemcpyDeviceToHost );
    if( err != hipSuccess ) {
        std::cout << "CUDA ERROR Memcpy->Host" << err << std::endl;
        hipFree(d);
        return;
    }
    hipFree(d);
}


void quicksort_gpu_par(std::vector<int> &list){
    // Detect settings
    // Copy to device
    int *d{};
    auto err = hipMalloc((void**)&d, sizeof(int) * list.size() );
    if( err != hipSuccess ) {
        std::cout << "CUDA ERROR Malloc" << err << std::endl;
        return;
    }
    err = hipMemcpy((void**)d, list.data(), sizeof(int) * list.size(), hipMemcpyHostToDevice );
    if( err != hipSuccess ) {
        std::cout << "CUDA ERROR Memcpy->Device" << err << std::endl;
        hipFree(d);
        return;
    }
    err = hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 24);
    if( err != hipSuccess ){
        std::cout << "CUDA ERROR set cudaLimitDevRuntimeSyncDepth" << std::endl;
        return;
    }

    quicksort_gpu_dyn_worker<<<1,1>>>(d, 0, list.size() - 1, 0);
    hipDeviceSynchronize();

    err = hipMemcpy(list.data(), (void**)d, sizeof(int) * list.size(), hipMemcpyDeviceToHost );
    if( err != hipSuccess ) {
        std::cout << "CUDA ERROR Memcpy->Host" << err << std::endl;
        hipFree(d);
        return;
    }
    hipFree(d);
}


///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// \brief TILE_WIDTH
///
/// //////////////////////////////
// Convenience function for printing lists of items in a vector
template <typename T>
std::ostream& operator<<(std::ostream& out, const std::vector<T> &list){

    bool first = true;
    for( const auto &item: list ){
        if( !first ) out << ", ";
        else first = false;
        out << item;
    }
    return out;
}

constexpr unsigned int TILE_WIDTH = 32;
template <typename T>
__global__
void ss_worker( T* data, size_t * stack, unsigned int size){
    auto idx = blockIdx.x * TILE_WIDTH + threadIdx.x;
    if( idx >= size ) return;

    size_t low  = stack[idx * 2];
    size_t high = stack[idx * 2 + 1];
    selection_sort_gpu(data, low, high);
}

template <typename T>
__global__
void qs_worker( T* data, size_t * qs_stack, size_t * qs_result_stack, size_t * ss_stack, unsigned int *qs_stack_size, unsigned int *ss_stack_size, unsigned int size){
    // thread_id to access the stack
    // get bounds from stack
    auto idx = blockIdx.x * TILE_WIDTH + threadIdx.x;

    // guard against too many threads
    if( idx >= size ) return;

    size_t low  = qs_stack[idx * 2];
    size_t high = qs_stack[idx * 2 + 1];
    // partition
    auto lh = partition_gpu( data, low, high);
    // if left partition okay,
        // atomicadd stack size (get this result)
        // atomicadd function returns old value
        // push to result stack

    if( low < lh.second ){
        // Down to last MIN_SIZE
        if( lh.second - low < MIN_SIZE ){
            int idx = atomicAdd(ss_stack_size, 1);
            ss_stack[idx * 2] = low;
            ss_stack[idx * 2 + 1] = lh.second;
        }else{
            int idx = atomicAdd(qs_stack_size, 1);
            qs_result_stack[idx * 2] = low;
            qs_result_stack[idx * 2 + 1] = lh.second;
        }
    }
    // if right partition okay,
        // atomicadd stack size (get this result)
        // atomicadd function returns old value
        // push to result stack

    if( lh.first < high ){
        // Down to last MIN_SIZE
        if( high - lh.first < MIN_SIZE ){
            int idx = atomicAdd(ss_stack_size, 1);
            ss_stack[idx * 2] = lh.first;
            ss_stack[idx * 2 + 1] = high;
        }else{
            int idx = atomicAdd(qs_stack_size, 1);
            qs_result_stack[idx * 2] = lh.first;
            qs_result_stack[idx * 2 + 1] = high;
        }
    }
}

template <typename T>
void quicksort_cpu_coordinated(std::vector<T> &list){
    if( list.empty() ) return;

    // data
    T *d{nullptr};

    // qs stack
    size_t * qs_stack{nullptr};
    // qs stack size
    unsigned int * qs_stack_size{nullptr};
    // qs result stack
    size_t * qs_result_stack{nullptr};

    // selection sort stack
    size_t * ss_stack{nullptr};
    // ss stack size
    unsigned int * ss_stack_size{nullptr};
    // ss worker stack
    size_t * ss_worker_stack{nullptr};

    // local sizes
    unsigned int h_qs = 1;
    unsigned int h_ss = 0;
    // prime the stack

    auto err = hipMalloc(reinterpret_cast<void**>(&qs_stack_size), sizeof(unsigned int));
         err = hipMalloc(reinterpret_cast<void**>(&ss_stack_size), sizeof(unsigned int));
         err = hipMalloc( reinterpret_cast<void**>(&qs_stack),        sizeof(size_t) * 2);
         err = hipMalloc( reinterpret_cast<void**>(&qs_result_stack), sizeof(size_t) * 4);
         err = hipMalloc(reinterpret_cast<void**>(&ss_stack),         sizeof(size_t) * 4);
         err = hipMalloc(reinterpret_cast<void**>(&d),                sizeof(T) * list.size() );

    if( err != hipSuccess ){
        std::cout << "CUDA ERROR getting memory" << std::endl;
        return;
    }

    err = hipMemcpy(reinterpret_cast<void**>(qs_stack_size), reinterpret_cast<void**>(&h_qs), sizeof(unsigned int), hipMemcpyHostToDevice);
    err = hipMemcpy(reinterpret_cast<void**>(ss_stack_size), reinterpret_cast<void**>(&h_ss), sizeof(unsigned int), hipMemcpyHostToDevice);
    err = hipMemcpy(reinterpret_cast<void**>(d), list.data(), sizeof(T)*list.size(), hipMemcpyHostToDevice);

    std::vector<size_t> init{0, list.size()-1};
    err = hipMemcpy(reinterpret_cast<void**>(qs_result_stack), init.data(), sizeof(size_t) * 2, hipMemcpyHostToDevice);

    if( err != hipSuccess ){
        std::cout << "CUDA ERROR copying memory" << std::endl;
        return;
    }

    while( h_qs || h_ss ){ // qs stack size || ss stack size > 0
        // if ss stack size > 0
            // swap ss worker stack and ss stack
            // create new stack for qs ? what is this for?
            // launch ss worker with ss worker stack

        if( h_ss ){
            // loads the ss_stack into ss_worker_stack
            std::swap( ss_worker_stack, ss_stack);
            // launch kernel for selection sort
            ss_worker<<<(h_ss + TILE_WIDTH - 1 )/TILE_WIDTH, TILE_WIDTH >>>(d, ss_worker_stack, h_ss );

            h_ss = 0;
            err = hipMemcpy(reinterpret_cast<void**>(ss_stack_size), reinterpret_cast<void**>(&h_ss), sizeof(unsigned int), hipMemcpyHostToDevice);
        }
        // free ss stack ( this was either unused last round, or are finished)
        hipFree(ss_stack);
        ss_stack = nullptr;


        // if qs stack size > 0
            // allocate ss stack with (qs stack size * 2)  to ensure if all reach threshold at same time we can accomadate
            // swap qs_stack and qs_result_stack
            // allocate result stack to qs stack size * 2
            // launch kernel

        if( h_qs ){
            size_t size = h_qs;
            h_qs = 0;
            hipMalloc(reinterpret_cast<void**>(&ss_stack), sizeof(size_t) * size * 2 );
            std::swap(qs_stack, qs_result_stack);
            hipFree(qs_result_stack);
            err = hipMalloc(reinterpret_cast<void**>(&qs_result_stack), sizeof(size_t) * size * 2 );
            if( err != hipSuccess ){
                std::cout << "CUDA ERROR getting memory in loop" << std::endl;
                return;
            }

            err = hipMemcpy(reinterpret_cast<void**>(qs_stack_size), reinterpret_cast<void**>(&h_qs), sizeof(unsigned int), hipMemcpyHostToDevice);
            // launch kernel
            qs_worker<<<(size + TILE_WIDTH - 1)/TILE_WIDTH, TILE_WIDTH >>>(d, qs_stack, qs_result_stack, ss_stack, qs_stack_size, ss_stack_size, size);
        }

        hipDeviceSynchronize();

        err = hipMemcpy(reinterpret_cast<void**>(&h_qs), reinterpret_cast<void**>(qs_stack_size), sizeof(unsigned int), hipMemcpyDeviceToHost);
        err = hipMemcpy(reinterpret_cast<void**>(&h_ss), reinterpret_cast<void**>(ss_stack_size), sizeof(unsigned int), hipMemcpyDeviceToHost);
    }

    err = hipMemcpy(list.data(), (void**)d, sizeof(T) * list.size(), hipMemcpyDeviceToHost );
    // deallocate spaces
    hipFree(qs_stack_size);
    hipFree(qs_stack);
    hipFree(qs_result_stack);
    hipFree(ss_stack);
    hipFree(ss_stack_size);
    hipFree(ss_worker_stack);
}
__host__
void initCuda(){hipFree(0);}


template void quicksort_gpu_dyn<int>(std::vector<int> &list);
template void quicksort_gpu_dyn<float>(std::vector<float> &list);
template void quicksort_cpu_coordinated(std::vector<int> &list);
template void quicksort_cpu_coordinated(std::vector<float> &list);
