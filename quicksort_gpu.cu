#include "hip/hip_runtime.h"
#include "quicksort_gpu.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <iostream>

template <typename T>
__device__
void swap_d(T& lhs, T& rhs){
    T tmp = lhs;
    lhs = rhs;
    rhs = tmp;
}

__device__
size_t partition_gpu( int* d, int low, int high){
    auto pivot = d[high];

    int i = low - 1;

    for( int j = low; j <= high - 1; ++j){
        if( d[j] < pivot ){
            ++i;
            swap_d(d[j], d[i]);
        }
    }

    swap_d( d[i+1], d[high]);
    return i + 1;
}

__global__
void quicksort_gpu_dyn_worker( int* d, int low, int high ){
    if( high <= low ) return;

    auto p = partition_gpu( d, low, high );

    hipStream_t s_l, s_h;
    hipStreamCreateWithFlags(&s_l, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&s_h, hipStreamNonBlocking);
    quicksort_gpu_dyn_worker<<<1, 1, 0, s_l>>>(d, low, p - 1);
    quicksort_gpu_dyn_worker<<<1, 1, 0, s_h>>>(d, p + 1, high );

    hipStreamDestroy(s_l);
    hipStreamDestroy(s_h);
}

__host__
void quicksort_gpu_dyn(std::vector<int> &list){
    // Detect settings
    // Copy to device
    int *d{};
    auto err = hipMalloc((void**)&d, sizeof(int) * list.size() );
    if( err != hipSuccess ) {
        std::cout << "CUDA ERROR Malloc" << err << std::endl;
        return;
    }
    err = hipMemcpy((void**)d, list.data(), sizeof(int) * list.size(), hipMemcpyHostToDevice );
    if( err != hipSuccess ) {
        std::cout << "CUDA ERROR Memcpy->Device" << err << std::endl;
        hipFree(d);
        return;
    }
    quicksort_gpu_dyn_worker<<<1,1>>>(d, 0, list.size() - 1);
    err = hipMemcpy(list.data(), (void**)d, sizeof(int) * list.size(), hipMemcpyDeviceToHost );
    if( err != hipSuccess ) {
        std::cout << "CUDA ERROR Memcpy->Host" << err << std::endl;
        hipFree(d);
        return;
    }
    hipFree(d);
    // Launch kernel
    // Copy to host
}


void quicksort_gpu_par(std::vector<int> &list){
    // Detect settings
    // Copy to device
    // Launch kernel
    // Copy to host
}

__host__
void initCuda(){hipFree(0);}
