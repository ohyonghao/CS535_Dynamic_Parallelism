#include "hip/hip_runtime.h"
#include "quicksort_gpu.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

template <typename T>
__device__
void swap_d(T& lhs, T& rhs){
    T tmp = lhs;
    lhs = rhs;
    rhs = tmp;
}

__device__
size_t partition_gpu_par( int* d, int low, int high){
    auto pivot = d[high];

    int i = low - 1;

    for( int j = low; j <= high - 1; ++j){
        if( d[j] < pivot ){
            ++i;
            swap_d(d[j], d[i]);
        }
    }

    swap_d( d[i+1], d[high]);
    return i + 1;
}

__global__
void quicksort_gpu_par_worker( int* d, int low, int high ){
    if( high >= low ) return;

    auto p = partition_gpu_par( d, low, high );

    hipStream_t s_l, s_h;
    hipStreamCreateWithFlags(&s_l, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&s_h, hipStreamNonBlocking);
    quicksort_gpu_par_worker<<<1, 1, 0, s_l>>>(d, low, p - 1);
    quicksort_gpu_par_worker<<<1, 1, 0, s_h>>>(d, p + 1, high );

    hipStreamDestroy(s_l);
    hipStreamDestroy(s_h);
}

__host__
void quicksort_gpu_par(std::vector<int> &list){
    // Detect settings
    // Copy to device
    int *d;
    hipMalloc((void**)d, sizeof(int) * list.size() );
    quicksort_gpu_par_worker<<<1,1>>>(d, 0, list.size() - 1);
    hipMemcpy((void**)d, list.data(), sizeof(int) * list.size(), hipMemcpyHostToDevice );
    hipFree(d);
    // Launch kernel
    // Copy to host
}


void quicksort_gpu_dyn(std::vector<int> &list){
    // Detect settings
    // Copy to device
    // Launch kernel
    // Copy to host
}

__host__
void initCuda(){hipFree(0);}
